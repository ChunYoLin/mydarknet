#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include "gpu_lane_detection.cpp"
#include "brake_light_gpu.cpp"
extern "C" {
#include "image.h"
#include "network.h"
#include "detection_layer.h"
#include "cost_layer.h"
#include "utils.h"
#include "parser.h"
#include "box.h"
#include "thpool.h"
#include <sys/time.h>
#include <signal.h>
#include <unistd.h>
}

/* Change class number here */
#define CLS_NUM 20

#ifdef OPENCV
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
extern "C" IplImage* image_to_Ipl(image img, int w, int h, int depth, int c, int step);
extern "C" image ipl_to_image(IplImage* src);
extern "C" void convert_yolo_detections(float *predictions, int classes, int num, int square, int side, int w, int h, float thresh, float **probs, box *boxes, int only_objectness);
extern "C" void draw_yolo(image im, int num, float thresh, box *boxes, float **probs);
extern "C" char *voc_names[];
extern "C" image voc_labels[];
extern "C" void draw_text(image a, char Text[], CvPoint TextPos);
#define BUFFERSIZE 300
#define step_m 1
#define step_op1 1
#define step_op2 1
#define ngetc(c) (read (0, (c), 1))
#define RESULT_SIZE 55
int result[RESULT_SIZE] = {0};
int current = 0;
static float **probs;
static box *boxes;
static network net;
static image in   ;

static cv::VideoCapture cap;
static cv::VideoWriter cap_out;
static float fps = 0;
static float demo_thresh = 0;
//static int w, h, depth, c, step= 0;
//static int MODE = -1;
timer_t timer_fetch,timer_m,timer_op1,timer_op2;



typedef struct ObjDetArg{
    int frameid;
    int draw;
}ODA;
typedef struct Ela_frame{
    pthread_mutex_t rwmutex;
    int frameid;
    image wholeframe;
	box_adjusted box_detected[10];
	int box_detected_num; 
	//IplImage *wholeframe_ipl;
}Eframe;
typedef struct Ela_frame_state{
    pthread_mutex_t fetch_mutex;
	pthread_mutex_t draw_mmutex;
	pthread_mutex_t draw_op1mutex;
	pthread_mutex_t draw_op2mutex;
	pthread_cond_t fetch_cond;
	pthread_cond_t draw_mcond;
	pthread_cond_t draw_op1cond;
	pthread_cond_t draw_op2cond;
    volatile int fetch;
    volatile int draw_m;
    volatile int draw_op1;
    volatile int draw_op2;
}Eframe_s;
//global variable declare
volatile int mode = 0;
threadpool thpool_cpu = thpool_init(1);
threadpool thpool_gpu = thpool_init(1);
//Eframe *frame_buffer = (Eframe*)malloc(sizeof(Eframe)*BUFFERSIZE);
Eframe frame_buffer[BUFFERSIZE] = {0};
Mat frame_buffer_m[BUFFERSIZE];
Eframe_s frame_buffer_s[2000] = {0};
//static const Eframe zeroEframe;
volatile int current_fetch_id = 0;
volatile int current_m_id = 0;
volatile int current_op1_id = 0;
volatile int current_op2_id = 0;
volatile int current_draw_id = 0;
float *X;
float *predictions;
pthread_mutex_t mutex;
pthread_mutex_t mutex_mode;
pthread_mutex_t mutex_current_fetch;
pthread_mutex_t mutex_current_m;
pthread_mutex_t mutex_current_op1;
pthread_mutex_t mutex_current_op2;
pthread_mutex_t mutex_current_draw;
pthread_mutex_t mutex_fetchjob;
pthread_mutex_t mutex_detect_mjob;
pthread_mutex_t mutex_detect_op1job;
pthread_mutex_t mutex_detect_op2job;
pthread_cond_t cond;
static CvCapture *capture;
//IplImage *frame;
struct timeval tval_before, tval_after, tval_result,tv;
struct timezone tz;
int s = 0;
void *fetch_in_thread(void *Elastic){
    gettimeofday(&tv,NULL);
    pthread_mutex_lock(&mutex_current_fetch);
    //printf("time: %ld ",tv.tv_usec/1000);
    //printf("start fetch frame %d \n",current_fetch_id);
    pthread_mutex_unlock(&mutex_current_fetch);
    cv::Mat frame_m;
    //frame_buffer[current_fetch_id%BUFFERSIZE].wholeframe_mat = frame_m;
	cap >> frame_m;
	gettimeofday(&tv,NULL);
	//printf("time: %ld ",tv.tv_usec/1000);
	//printf("start detect frame %d's lane\n",current_fetch_id);

	int output = gpu_lane_detection(frame_m,result,RESULT_SIZE,current);
	gettimeofday(&tv,NULL); 
    //printf("time: %ld ",tv.tv_usec/1000);
	//printf("finish detect frame %d's lane \n",current_fetch_id);
    IplImage frame = frame_m;
	//frame = cvQueryFrame(capture);
    //frame_buffer[current_fetch_id%BUFFERSIZE].wholeframe_ipl = &frame;
	//cvSaveImage("foo2.png",&frame_buffer[current_fetch_id%BUFFERSIZE].wholeframe_ipl);
	in = ipl_to_image(&frame);
    rgbgr_image(in);
    free_image(frame_buffer[current_fetch_id%BUFFERSIZE].wholeframe);
    frame_buffer[current_fetch_id%BUFFERSIZE].wholeframe = in;
    frame_buffer_m[current_fetch_id%BUFFERSIZE] = frame_m;
    frame_buffer[current_fetch_id%BUFFERSIZE].frameid = current_fetch_id;
    gettimeofday(&tv,NULL); 
    //printf("time: %ld ",tv.tv_usec/1000);
    //printf("finish fetch frame %d \n",current_fetch_id);
	
    frame_buffer_s[current_fetch_id].fetch = 1;

    current_fetch_id++;
    return 0;
}

void *detect_in_thread(void *arg)
{
    
    ODA tmp = *((ODA*)arg);
    image ROI;
    Eframe *detectframe;
	Mat detectframe_m;
	while(!frame_buffer_s[tmp.frameid].fetch);
    detectframe = &frame_buffer[tmp.frameid%BUFFERSIZE];
	detectframe_m = frame_buffer_m[tmp.frameid%BUFFERSIZE];
	gettimeofday(&tv,&tz);
    //printf("time: %ld ",tv.tv_usec/1000);
    if(tmp.draw == 0){
		ROI = detectframe->wholeframe;
    }
    else if(tmp.draw == 1){
		ROI = crop_image(detectframe->wholeframe,886,560,448,448);
		//printf("start detect frame %d's mandatory \n",tmp.frameid);
    }
    else if(tmp.draw == 2){
		ROI = crop_image(detectframe->wholeframe,438,560,448,448);
    	//printf("start detect frame %d's optional1 \n",tmp.frameid);
    }
    else if(tmp.draw == 3){
		ROI = crop_image(detectframe->wholeframe,1334,560,448,448);
		//printf("start detect frame %d's optional2 \n",tmp.frameid);
    }
    float nms = .4;
    detection_layer l = net.layers[net.n-1];
    X = ROI.data;
    predictions = network_predict(net, X);
    free_image(ROI);
    convert_yolo_detections(predictions, l.classes, l.n, l.sqrt, l.side, 1, 1, demo_thresh, probs, boxes, 0);
    if (nms > 0) do_nms(boxes, probs, l.side*l.side*l.n, l.classes, nms);
	memset(detectframe->box_detected, 0, sizeof(detectframe->box_detected));
    //draw_detections(detectframe->wholeframe, l.side*l.side*l.n, demo_thresh, boxes, probs, voc_names, voc_labels, CLS_NUM,tmp.draw);
	detectframe->box_detected_num = cal_boxdetected_info(detectframe->wholeframe, l.side*l.side*l.n, demo_thresh, boxes, probs, voc_names, voc_labels, CLS_NUM, tmp.draw, detectframe->box_detected);
	for(int i = 0;i < detectframe->box_detected_num;i++){	
		int x = detectframe->box_detected[i].left;
		int y = detectframe->box_detected[i].top;
		int h = detectframe->box_detected[i].bot - detectframe->box_detected[i].top + 1;
		int w = detectframe->box_detected[i].right - detectframe->box_detected[i].left + 1;
		Rect region_of_interest = Rect(x, y, w, h);
		printf("x:%d y:%d w:%d h:%d\n",x,y,w,h);
		cv::Mat car = detectframe_m(region_of_interest);
		bool warning = Brake_light(car);
		printf("warning : %d\n",warning);
	}
	gettimeofday(&tv,&tz);
    //printf("time: %ld ",tv.tv_usec/1000);
    if(tmp.draw == 1){
		pthread_mutex_lock(&frame_buffer_s[tmp.frameid].draw_mmutex);
		frame_buffer_s[tmp.frameid].draw_m = 1;
		pthread_mutex_unlock(&frame_buffer_s[tmp.frameid].draw_mmutex);
		//printf("finish detect frame %d's mandatory \n",tmp.frameid);
		pthread_mutex_lock(&mutex_current_m);
		current_m_id+=step_m;
		pthread_mutex_unlock(&mutex_current_m);
    }
    else if(tmp.draw == 2){
		pthread_mutex_lock(&frame_buffer_s[tmp.frameid].draw_op1mutex);
		frame_buffer_s[tmp.frameid].draw_op1 = 1;
		pthread_mutex_unlock(&frame_buffer_s[tmp.frameid].draw_op1mutex);
		//printf("finish detect frame %d's optional1 \n",tmp.frameid);
		pthread_mutex_lock(&mutex_current_op1);
		current_op1_id+=step_op1;
		pthread_mutex_unlock(&mutex_current_op1);
    }
    else if(tmp.draw == 3){
		pthread_mutex_lock(&frame_buffer_s[tmp.frameid].draw_op2mutex);
		frame_buffer_s[tmp.frameid].draw_op2 = 1;
		pthread_mutex_unlock(&frame_buffer_s[tmp.frameid].draw_op2mutex);
    	//printf("finish detect frame %d's optional2 \n",tmp.frameid);
		pthread_mutex_lock(&mutex_current_op2);
		current_op2_id+=step_op2;
		pthread_mutex_unlock(&mutex_current_op2);
    }
    //pthread_cond_signal(&cond);
    //print FPS
    //printf("\033[2J");
    //printf("\033[1;1H");
    //printf("\nFPS:%.0f\n",fps);
    //printf("Objects:\n\n");

    return 0;
}

void *show_frame(void *arg){
    while(1){
		pthread_mutex_lock(&mutex_current_draw);
		int tmpdrawid = current_draw_id;
		pthread_mutex_unlock(&mutex_current_draw);
		pthread_mutex_lock(&mutex_mode);
		int tmpmode = mode;
    	pthread_mutex_unlock(&mutex_mode); 
        if(tmpmode == 0){
	    	while(!frame_buffer_s[tmpdrawid].fetch){			
				if(tmpdrawid % step_m == 0){
					//while(!frame_buffer_s[tmpdrawid].draw_m){
						//printf("stuck in frame %d draw_m\n",tmpdrawid);
					//	;
					//}
				}
				if(tmpdrawid % step_op1 == 0){
					//while(!frame_buffer_s[tmpdrawid].draw_op1){
						//printf("stuck in frame %d draw_op1\n",tmpdrawid);
						//;
					//}
				}
				if(tmpdrawid % step_op2 == 0){
					//while(!frame_buffer_s[tmpdrawid].draw_op2){
						//printf("stuck in frame %d draw_op2\n",tmpdrawid);
						//;
					//}
				}
			}
	    	draw_box(frame_buffer[tmpdrawid%BUFFERSIZE].wholeframe,886,560,1334,1008+1,0,0,255);
   	    	draw_box(frame_buffer[tmpdrawid%BUFFERSIZE].wholeframe,438-1,560,886,1008+1,255,0,0);
    		draw_box(frame_buffer[tmpdrawid%BUFFERSIZE].wholeframe,1334,560,1782+1,1008+1,255,0,0);
    		gettimeofday(&tv,&tz);
    		//printf("time: %ld ",tv.tv_usec/1000);
	   		//printf("finish drawing frame %d \n",current_draw_id);
	    	show_image(frame_buffer[tmpdrawid%BUFFERSIZE].wholeframe,"YOLO");			
	    	cvWaitKey(1);
			pthread_mutex_lock(&mutex_current_draw);
	    	current_draw_id++;
			pthread_mutex_unlock(&mutex_current_draw);
		}
    }
}
void timerHandler( int sig, siginfo_t *si, void *uc ){
    timer_t *tidp;
    tidp = (timer_t *)si->si_value.sival_ptr;
    ODA *tmp = (ODA*)malloc(sizeof(ODA));
    static int fetchjobnum = 0;
    static int detect_mjobnum = 0;
    static int detect_op1jobnum = 0;
    static int detect_op2jobnum = 0;
    if ( *tidp == timer_fetch ){
		pthread_mutex_lock(&mutex_fetchjob);
		gettimeofday(&tv,&tz);
    	//printf("time: %d ",tv.tv_usec/1000);
		//printf("add work fetch frame %d\n",fetchjobnum);
		thpool_add_work(thpool_cpu,fetch_in_thread,0);
		fetchjobnum++;
		pthread_mutex_unlock(&mutex_fetchjob);
    }
    else if ( *tidp == timer_m ){
		pthread_mutex_lock(&mutex_current_fetch);
        tmp->frameid = current_fetch_id;
        pthread_mutex_unlock(&mutex_current_fetch);
		//pthread_mutex_lock(&mutex_detect_mjob);
		//tmp->frameid = detect_mjobnum;
		tmp->draw = 1;
		gettimeofday(&tv,&tz);
    	//printf("time: %ld ",tv.tv_usec/1000);
		//printf("add work detect frame %d's mandatory \n",tmp->frameid);
        thpool_add_work(thpool_gpu,detect_in_thread,tmp);
		//detect_mjobnum+=step_m;
		//pthread_mutex_unlock(&mutex_detect_mjob);
    }
    else if ( *tidp == timer_op1 ){
		pthread_mutex_lock(&mutex_current_fetch);
		tmp->frameid = current_fetch_id;
		pthread_mutex_unlock(&mutex_current_fetch);
		//pthread_mutex_lock(&mutex_detect_op1job);
		//tmp->frameid = detect_op1jobnum;
		tmp->draw = 2;
		gettimeofday(&tv,&tz);
    	//printf("time: %ld ",tv.tv_usec/1000);
		//printf("add work detect frame %d's optional1 \n",tmp->frameid);
        thpool_add_work(thpool_gpu,detect_in_thread,tmp);
		//detect_op1jobnum+=step_op1;
		//pthread_mutex_unlock(&mutex_detect_op1job);
    }
    else if ( *tidp == timer_op2 ){
		pthread_mutex_lock(&mutex_current_fetch);
		tmp->frameid = current_fetch_id;
		pthread_mutex_unlock(&mutex_current_fetch);
		//pthread_mutex_lock(&mutex_detect_op2job);
		//tmp->frameid = detect_op2jobnum;
		tmp->draw = 3;
		gettimeofday(&tv,&tz);
    	//printf("time: %ld ",tv.tv_usec/1000);
		//printf("add work detect frame %d's optional2 \n",tmp->frameid);
        thpool_add_work(thpool_gpu,detect_in_thread,tmp);
		//detect_op2jobnum+=step_op2;
		//pthread_mutex_unlock(&mutex_detect_op2job);
    }
}
int makeTimer( timer_t *timerID, int expireMS, int intervalMS ){
    struct sigevent te;
    struct itimerspec its;
    struct sigaction sa;
    int sigNo = SIGRTMIN;
    /* Set up signal handler. */
    sa.sa_flags = SA_SIGINFO;
    sa.sa_sigaction = timerHandler;
    sigemptyset(&sa.sa_mask);
    if (sigaction(sigNo, &sa, NULL) == -1){
        printf("error");
    }
    /* Set and enable alarm */
    te.sigev_notify = SIGEV_SIGNAL;
    te.sigev_signo = sigNo;
    te.sigev_value.sival_ptr = timerID;
    timer_create(CLOCK_REALTIME, &te, timerID);
    its.it_interval.tv_sec = 0;
    its.it_interval.tv_nsec = intervalMS * 1000000;
    its.it_value.tv_sec = 0;
    its.it_value.tv_nsec = expireMS * 1000000;
    timer_settime(*timerID, 0, &its, NULL);
    timer_gettime(*timerID,&its);
    return(0);
}

void *TIMER(void *arg){
    gettimeofday(&tv,&tz);
    //printf("time: %ld ",tv.tv_usec/1000);
    //printf("start all timer\n");
    if(mode == 0){
		makeTimer(&timer_fetch, 33, 33);
    	makeTimer(&timer_m, 100, 100);
		//makeTimer(&timer_op1, 200, 200);
    	//makeTimer(&timer_op2, 200, 200);
    }
    else if(mode == 1){
    	makeTimer(&timer_fetch, 33, 33);
		makeTimer(&timer_m, 100, 100);
    }
    while(1);
}
void *MODE_CONTROLLER(void *arg){
    char c;
    while(1){
        ngetc(&c);
		if(c != '\n'){
			printf("c == %c\n",c);
			pthread_mutex_lock(&mutex_mode);
			if(c == '0'){
			    printf("mode0\n");
			    mode = 0;
			}
			else if(c == '1'){
			    printf("mode1\n");
			    mode = 1;
			}
			else if(c == '2'){
				printf("mode2\n");
				mode = 2;
			}
			pthread_mutex_unlock(&mutex_mode);
			pthread_cond_signal(&cond);
			printf("signal\n");
        }
    }
}

extern "C" void demo_yolo(char *cfgfile, char *weightfile, float thresh, int cam_index, char *videofile){
    demo_thresh = thresh;
    printf("YOLO demo\n");
    net = parse_network_cfg(cfgfile);
    if(weightfile){
        load_weights(&net, weightfile);
    }
    set_batch_network(&net, 1);
    srand(2222222);

if(cam_index != -1){
    //MODE = 0; 
    cv::VideoCapture cam(cam_index);
    cap = cam;
    if(!cap.isOpened()) error("Couldn't connect to webcam.\n");
}
else{
    //MODE = 1;
    printf("Video File name is: %s\n", videofile);
    capture = cvCreateFileCapture(videofile);
    cv::VideoCapture videoCap(videofile);
    
    cap = videoCap;
    if(!cap.isOpened()) error("Couldn't read video file.\n");
}
	//initial every mutex
    for(int i = 0;i < BUFFERSIZE;i++)pthread_mutex_init(&frame_buffer[i].rwmutex,NULL);
    for(int i = 0;i < 2000;i++){
		pthread_mutex_init(&frame_buffer_s[i].fetch_mutex,NULL);
		pthread_mutex_init(&frame_buffer_s[i].draw_mmutex,NULL);
		pthread_mutex_init(&frame_buffer_s[i].draw_op1mutex,NULL);
		pthread_mutex_init(&frame_buffer_s[i].draw_op2mutex,NULL);
		pthread_cond_init(&frame_buffer_s[i].fetch_cond,NULL);
		pthread_cond_init(&frame_buffer_s[i].draw_mcond,NULL);
		pthread_cond_init(&frame_buffer_s[i].draw_op1cond,NULL);
		pthread_cond_init(&frame_buffer_s[i].draw_op2cond,NULL);
	}
    pthread_mutex_init(&mutex,NULL);
    pthread_mutex_init(&mutex_mode,NULL);
    pthread_mutex_init(&mutex_current_fetch,NULL);
    pthread_mutex_init(&mutex_current_m,NULL);
    pthread_mutex_init(&mutex_current_op1,NULL);
    pthread_mutex_init(&mutex_current_op2,NULL);
	pthread_mutex_init(&mutex_current_draw,NULL);
    pthread_mutex_init(&mutex_fetchjob,NULL);
    pthread_mutex_init(&mutex_detect_mjob,NULL);
    pthread_mutex_init(&mutex_detect_op1job,NULL);
    pthread_mutex_init(&mutex_detect_op2job,NULL);
    pthread_cond_init(&cond,NULL);

    detection_layer l = net.layers[net.n-1];
    boxes = (box *)calloc(l.side*l.side*l.n, sizeof(box));
    probs = (float **)calloc(l.side*l.side*l.n, sizeof(float *));
    for(int j = 0; j < l.side*l.side*l.n; ++j) probs[j] = (float *)calloc(l.classes, sizeof(float *));
    pthread_t timer,drawer,modecontroller;
    ODA *arg = (ODA*)malloc(sizeof(ODA));
    pthread_create(&modecontroller,0,MODE_CONTROLLER,0);
    while(1){
		pthread_cond_wait(&cond,&mutex);
        pthread_create(&timer,0,TIMER,0);
        pthread_create(&drawer,0,show_frame,0);
    }
    //while(1);
}
#else
extern "C" void demo_yolo(char *cfgfile, char *weightfile, float thresh, int cam_index){
    fprintf(stderr, "YOLO demo needs OpenCV for webcam images.\n");
}
#endif

