#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "network.h"
#include "detection_layer.h"
#include "cost_layer.h"
#include "utils.h"
#include "parser.h"
#include "box.h"
#include "image.h"
#include <sys/time.h>
}

/* Change class number here */
#define CLS_NUM 20

#ifdef OPENCV
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
extern "C" IplImage* image_to_Ipl(image img, int w, int h, int depth, int c, int step);
extern "C" image ipl_to_image(IplImage* src);
extern "C" void convert_yolo_detections(float *predictions, int classes, int num, int square, int side, int w, int h, float thresh, float **probs, box *boxes, int only_objectness);
extern "C" void draw_yolo(image im, int num, float thresh, box *boxes, float **probs);

extern "C" char *voc_names[];
extern "C" image voc_labels[];
extern "C" void draw_text(image a, char Text[], CvPoint TextPos);
static float **probs;
static box *boxes;
static network net;
static image in   ;
static image in_s ;
static image in_op1;
static image in_op2;
static image det  ;
static image det_s;
static image det_op1;
static image det_op2;
static image disp ;
static cv::VideoCapture cap;
static cv::VideoWriter cap_out;
static float fps = 0;
static float demo_thresh = 0;
static int w, h, depth, c, step= 0;
static int MODE = -1;
int Elastic = 1;
void *fetch_in_thread(void *ptr)
{
    cv::Mat frame_m;   
    cap >> frame_m;
    IplImage frame = frame_m;
    //mandatory
    cv::Mat frame_cropM;
    cv::Point M_p1(886,560);
    cv::Point M_p2(1334,1008);
    cv::Rect ROI_M(M_p1,M_p2);
    frame_cropM = frame_m(ROI_M).clone();
    cv::rectangle(frame_m,ROI_M,cv::Scalar(0,0,255),2);
    IplImage frame_ROIM = frame_cropM;
    //optional1
    cv::Mat frame_cropop1;
    cv::Point op1_p1(438,560);
    cv::Point op1_p2(886,1008);
    cv::Rect ROI_op1(op1_p1,op1_p2);
    frame_cropop1 = frame_m(ROI_op1).clone();
    cv::rectangle(frame_m,ROI_op1,cv::Scalar(255,0,0),2);
    IplImage frame_ROIop1 = frame_cropop1;
    //optional2
    cv::Mat frame_cropop2;
    cv::Point op2_p1(1334,560);
    cv::Point op2_p2(1782,1008);
    cv::Rect ROI_op2(op2_p1,op2_p2);
    frame_cropop2 = frame_m(ROI_op2).clone();
    cv::rectangle(frame_m,ROI_op2,cv::Scalar(255,0,0),2);
    IplImage frame_ROIop2 = frame_cropop2;
if(step == 0)
{
    w = frame.width;
    h = frame.height;
    c = frame.nChannels;
    depth= frame.depth; 
    step = frame.widthStep;
}   
    
    in = ipl_to_image(&frame);
    rgbgr_image(in);
    if(!Elastic)in_s = resize_image(in, net.w, net.h);
    else{
    	in_s = ipl_to_image(&frame_ROIM);
    	in_op1 = ipl_to_image(&frame_ROIop1);
    	in_op2 = ipl_to_image(&frame_ROIop2);
    	rgbgr_image(in_s);
    	rgbgr_image(in_op1);
    	rgbgr_image(in_op2);
    }
    return 0;
}

void *detect_in_thread(void *ptr)
{
    float nms = .4;
    if(!Elastic){
	detection_layer l = net.layers[net.n-1];
	float *X = det_s.data;
	float *predictions = network_predict(net, X);
	free_image(det_s);
        convert_yolo_detections(predictions, l.classes, l.n, l.sqrt, l.side, 1, 1, demo_thresh, probs, boxes, 0);
	if (nms > 0) do_nms(boxes, probs, l.side*l.side*l.n, l.classes, nms);
	draw_detections(det, l.side*l.side*l.n, demo_thresh, boxes, probs, voc_names, voc_labels, CLS_NUM,0);
    }
    else{
	//detect mandatory
	detection_layer l = net.layers[net.n-1];
	float *X = det_s.data;
	float *predictions = network_predict(net, X);
	free_image(det_s);
	convert_yolo_detections(predictions, l.classes, l.n, l.sqrt, l.side, 1, 1, demo_thresh, probs, boxes, 0);
	if (nms > 0) do_nms(boxes, probs, l.side*l.side*l.n, l.classes, nms);
	draw_detections(det, l.side*l.side*l.n, demo_thresh, boxes, probs, voc_names, voc_labels, CLS_NUM,1);
	//detect optional1
	X = det_op1.data;
	predictions = network_predict(net,X);
	free_image(det_op1);
	convert_yolo_detections(predictions, l.classes, l.n, l.sqrt, l.side, 1, 1, demo_thresh, probs, boxes, 0);
	if (nms > 0) do_nms(boxes, probs, l.side*l.side*l.n, l.classes, nms);
	draw_detections(det, l.side*l.side*l.n, demo_thresh, boxes, probs, voc_names, voc_labels, CLS_NUM,2);
	//detect optional2
	X = det_op2.data;
	predictions = network_predict(net,X);
	free_image(det_op2);
	convert_yolo_detections(predictions, l.classes, l.n, l.sqrt, l.side, 1, 1, demo_thresh, probs, boxes, 0);
	if (nms > 0) do_nms(boxes, probs, l.side*l.side*l.n, l.classes, nms);
	draw_detections(det, l.side*l.side*l.n, demo_thresh, boxes, probs, voc_names, voc_labels, CLS_NUM,3);
    }
    //print FPS
    printf("\033[2J");
    printf("\033[1;1H");
    printf("\nFPS:%.0f\n",fps);
    printf("Objects:\n\n");
    if(MODE == 1)
    {
        IplImage* outputIpl= image_to_Ipl(det, w, h, depth, c, step);
        cv::Mat outputMat = cv::cvarrToMat(outputIpl, true);
        /*
        cvNamedWindow("image", CV_WINDOW_AUTOSIZE);
        cvShowImage("image", outputIpl); 
        cvWaitKey(1);  
        */
        cvReleaseImage(&outputIpl);
        cap_out << outputMat;
        outputMat.release();
     }

    return 0;
}
extern "C" void demo_yolo(char *cfgfile, char *weightfile, float thresh, int cam_index, char *videofile)
{
    demo_thresh = thresh;
    printf("YOLO demo\n");
    net = parse_network_cfg(cfgfile);
    if(weightfile){
        load_weights(&net, weightfile);
    }
    set_batch_network(&net, 1);

    srand(2222222);

if(cam_index != -1)
{
    MODE = 0; 
    cv::VideoCapture cam(cam_index);
    cap = cam;
    if(!cap.isOpened()) error("Couldn't connect to webcam.\n");
}
else 
{
    MODE = 1;
    printf("Video File name is: %s\n", videofile);
    cv::VideoCapture videoCap(videofile);
    cap = videoCap;
    if(!cap.isOpened()) error("Couldn't read video file.\n");

    cv::Size S = cv::Size((int)videoCap.get(CV_CAP_PROP_FRAME_WIDTH), (int)videoCap.get(CV_CAP_PROP_FRAME_HEIGHT));
    cv::VideoWriter outputVideo("out.avi", CV_FOURCC('D','I','V','X'), videoCap.get(CV_CAP_PROP_FPS), S, true);
    if(!outputVideo.isOpened()) error("Couldn't write video file.\n");
    cap_out = outputVideo;
}
 
    detection_layer l = net.layers[net.n-1];
    int j;

    boxes = (box *)calloc(l.side*l.side*l.n, sizeof(box));
    probs = (float **)calloc(l.side*l.side*l.n, sizeof(float *));
    for(j = 0; j < l.side*l.side*l.n; ++j) probs[j] = (float *)calloc(l.classes, sizeof(float *));

    pthread_t fetch_thread;
    pthread_t detect_thread;
    fetch_in_thread(0);
    det = in;
    det_s = in_s;
    det_op1 = in_op1;
    det_op2 = in_op2;
    fetch_in_thread(0);
    detect_in_thread(0);
    disp = det;
    det = in;
    det_s = in_s;
    det_op1 = in_op1;
    det_op2 = in_op2;
    while(1){
        struct timeval tval_before, tval_after, tval_result;
        
        if(pthread_create(&fetch_thread, 0, fetch_in_thread, 0)) error("Thread creation failed");
	gettimeofday(&tval_before, NULL);
        if(pthread_create(&detect_thread, 0, detect_in_thread, 0)) error("Thread creation failed");
 	//if(pthread_create(&detect_thread_op1, 0, detect_in_thread_op1, 0)) error("Thread creation failed");
	show_image(disp, "YOLO");       
	free_image(disp);
        cvWaitKey(1);
        pthread_join(fetch_thread, 0);
        pthread_join(detect_thread, 0);
	//pthread_join(detect_thread_op1, 0);
        disp  = det;
        det   = in;
        det_s = in_s;
        det_op1 = in_op1;
        det_op2 = in_op2;
        gettimeofday(&tval_after, NULL);
        timersub(&tval_after, &tval_before, &tval_result);
        float curr = 1000000.f/((long int)tval_result.tv_usec);
        fps = .9*fps + .1*curr;
    }
}
#else
extern "C" void demo_yolo(char *cfgfile, char *weightfile, float thresh, int cam_index){
    fprintf(stderr, "YOLO demo needs OpenCV for webcam images.\n");
}
#endif

